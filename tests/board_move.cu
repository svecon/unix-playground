#include "hip/hip_runtime.h"
// CUDA implementation of random go player
// (c) Petr Baudis <pasky@ucw.cz>  2009


// FIXME: No ko detection
/* Actually, I don't think ko detection is so important; it will mud down
   playouts somewhat, but eventually all the moves to be made except one
   ko fight are made anyway and MAX_MOVES catches the last ko. */

#include <stdio.h>
#include <stdlib.h>
#include <sys/times.h>
#include <unistd.h>

#define S 11
#define S2 S*S
#define MAX_MOVES (S2 * 2)

struct board {
#define S_NONE 0
#define S_BLACK 1
#define S_WHITE 2
#define S_EDGE 3
	int stone[S2];

	/* >0: coordinate of "group center"
	 * 0: no stone there
	 * -1: eye forbidden for black to play
	 * -2: eye forbidden for white to play
	 * -3: eye forbidden for both to play */
	int group[S2];
	int libs[S2];

	float p[S2]; /* probability of play; sum = 1 */

	int random;
	int free_spots[2]; /* free spots # for black and white */
	int to_play;
	int moves;
	int komi; /* <0 black wins, >0 white wins */
} b;


__device__ unsigned int
pm_random(unsigned int seed)
{
	unsigned long hi, lo;
	lo = 16807 * (seed & 0xffff);
	hi = 16807 * (seed >> 16);
	lo += (hi & 0x7fff) << 16;
	lo += hi >> 15;
	seed = (lo & 0x7fffffff) + (lo >> 31);
	//return ((seed & 0xffff) * max) >> 16;
	return seed;
}


#define TL (threadIdx.x - 1)
#define TR (threadIdx.x + 1)
#define TU (threadIdx.x - S)
#define TD (threadIdx.x + S)

__device__ void
dprint_board(struct board *bp)
{
#if 0
	for (int i = 0; i < S; i++) {
		int j;
		for (j = 0; j < S; j++) {
			int st = bp->stone[i * S + j];
			printf("%c ", st == S_EDGE ? '#' : st == S_WHITE ? 'O' : st == S_BLACK ? 'X' : '.');
		}
		printf(" ");
		for (j = 0; j < S; j++)
			printf("%03d ", bp->group[i * S + j]);
		printf(" ");
		for (j = 0; j < S; j++)
			printf("%02d ", bp->libs[i * S + j]);
#if 0
		for (j = 0; j < S; j++)
			printf("%1.02f ", bp->p[i * S + j]);
#endif
		printf("\n");
	}
	printf("random %d free_spots %d,%d to_play %d\n", bp->random, bp->free_spots[0], bp->free_spots[1], bp->to_play);
#endif
}

__device__ void
update_libs(struct board *bp, int delta, int except)
{
	int groups[4] = { bp->group[TU], bp->group[TL], bp->group[TR], bp->group[TD] };
	/* A loop over groups[] is somehow never unrolled and groups[] is forced to
	 local memory */
	if (groups[0] != except)
		atomicAdd(&bp->libs[groups[0]], delta);
	if (groups[1] != except && groups[1] != groups[0])
		atomicAdd(&bp->libs[groups[1]], delta);
	if (groups[2] != except && groups[2] != groups[1] && groups[2] != groups[0])
		atomicAdd(&bp->libs[groups[2]], delta);
	if (groups[3] != except && groups[3] != groups[2] && groups[3] != groups[1] && groups[3] != groups[0])
		atomicAdd(&bp->libs[groups[3]], delta);
}

__device__ void
capture_stones(struct board *bp)
{
	bp->libs[threadIdx.x] = bp->libs[bp->group[threadIdx.x]];
	if (bp->libs[threadIdx.x] == 0) {
		update_libs(bp, +1, bp->group[threadIdx.x]);
		bp->stone[threadIdx.x] = S_NONE;
		bp->group[threadIdx.x] = 0;
		atomicAdd(&bp->free_spots[0], 1);
		atomicAdd(&bp->free_spots[1], 1);
	}
}

__device__ void
survey_eye(struct board &b)
{
	int nei[4] = {TU, TL, TR, TD};
	int dnei[4] = {TU-1, TU+1, TD-1, TD+1};
	int stonecount = 0; // each bye is one direction
	/* We have to manually do bit magic, nvcc is too stupid and
	 would force stonecount[] to local memory */
#define STONECOUNT_ANY(stone) (stonecount & (0xf << ((stone) * 4)))
#define STONECOUNT(stone) ((stonecount & (0xf << ((stone) * 4))) >> ((stone) * 4))
	for (int i = 0; i < 4; i++) {
		int s = b.stone[nei[i]];
		int sc = STONECOUNT(s) + 1;
		stonecount = stonecount & ~(0xf << (s * 4)) | (sc << (s * 4));
	}
	if (STONECOUNT_ANY(S_NONE) || (STONECOUNT_ANY(S_BLACK) && STONECOUNT_ANY(S_WHITE)))
		return;
	bool is_white = STONECOUNT_ANY(S_WHITE);
	bool on_edge = STONECOUNT_ANY(S_EDGE);

	/* False eyes aren't forbidden, however. */
	/* XXX: We don't support http://senseis.xmp.net/?TwoHeadedDragon */
	stonecount = 0;
	for (int i = 0; i < 4; i++) {
		int s = b.stone[dnei[i]];
		int sc = STONECOUNT(s) + 1;
		stonecount = stonecount & ~(0xf << (s * 4)) | (sc << (s * 4));
	}
	if (on_edge + STONECOUNT(is_white ? S_BLACK : S_WHITE) > 1) {
		/* This might've been forbidden eye in past - ponnuki */
		if (b.group[threadIdx.x] < 0) {
			if ((-b.group[threadIdx.x]) & S_BLACK)
				atomicAdd(&b.free_spots[0], 1);
			if ((-b.group[threadIdx.x]) & S_WHITE)
				atomicAdd(&b.free_spots[1], 1);
			b.group[threadIdx.x] = 0;
		}
		return;
	}

	/* In case of the last liberty, the other player can play in the eye. */
	bool last_lib = (b.stone[TL] != S_EDGE ? b.libs[TL] : b.libs[TR]) < 2;
	switch (b.group[threadIdx.x]) {
		case 0:
			/* Freshly appeared eye; remove it from the relevant pools */
			if (!last_lib || !is_white)
				atomicSub(&b.free_spots[0], 1);
			if (!last_lib || is_white)
				atomicSub(&b.free_spots[1], 1);
			break;
		case -1:
			/* Formerly half-forbidden eye; if not anymore, remove it from the other player's pool */
			if (!last_lib)
				atomicSub(&b.free_spots[1], 1);
			break;
		case -2:
			if (!last_lib)
				atomicSub(&b.free_spots[0], 1);
			break;
		case -3:
			/* Formerly forbidden eye; possibly allow one player to play inside */
			if (last_lib)
				atomicAdd(&b.free_spots[1 - is_white], 1);
			break;
	}
	b.group[threadIdx.x] = last_lib ? -1 - is_white : -3;
}

__device__ void
calc_probability(struct board &b)
{
	int group = b.group[threadIdx.x];
	if (b.stone[threadIdx.x] == S_NONE
	    && (group == 0
		/* eye forbidden just for the other player */
		|| (group < 0 && !((-group) & b.to_play)))) {
		b.p[threadIdx.x] = 1.F / b.free_spots[b.to_play - 1];
	} else {
		b.p[threadIdx.x] = 0;
	}
}

__device__ void
play_one_move(struct board &b)
{
	__shared__ int group_merge_n, group_merge[4];
	__shared__ int move;

	/** Get a random number */
	if (!threadIdx.x)
		b.random = pm_random(b.random);

	/** Choose a move to play. */
	/* So-called weighted random selection; build a tree of probability
	   bounds in O(logN), then check one node per thread. We don't need
	   to bother with downsweep-reduce, our array is fairly tiny. */
	/* upbound[] is double-buffered */
	__shared__ float upbound[S2 * 2];
	int outo = 0, ino = 1;
	upbound[threadIdx.x] = b.p[threadIdx.x];
	__syncthreads();
	for (int d = 1; d < S2; d *= 2) {
		outo = 1 - outo; ino = 1 - ino;
		if (threadIdx.x >= d)
			upbound[outo * S2 + threadIdx.x] = upbound[ino * S2 + threadIdx.x] + upbound[ino * S2 + threadIdx.x - d];
		else
			upbound[outo * S2 + threadIdx.x] = upbound[ino * S2 + threadIdx.x];
		__syncthreads();
	}

	/** Place the stone */

	float p = (1 + float(b.random & 0xffff)) / 65536;
	int to_play;
	// printf("[%d] %f < %f < %f\n", threadIdx.x, threadIdx.x ? upbound[outo * S2 + threadIdx.x - 1] : -1.F, p, upbound[outo * S2 + threadIdx.x]);
	if (p <= upbound[outo * S2 + threadIdx.x] && (!threadIdx.x || upbound[outo * S2 + threadIdx.x - 1] < p)) {
		move = threadIdx.x;
		b.stone[move] = b.to_play;
		/* Take off liberty from surrounding groups */
		update_libs(&b, -1, 0);
		group_merge_n = 0;
	} else {
		to_play = b.to_play;
	}
	__syncthreads();

	/** Survey if the stone can join existing group */
	switch (move - (int)threadIdx.x) {
		case 0:
			to_play = b.to_play == S_BLACK ? S_WHITE : S_BLACK;
			// XXX: Two ifs are probably more efficient than full branch
			if (b.group[threadIdx.x] < 0) {
				/* Half-forbidden eye */
				atomicSub(&b.free_spots[b.to_play - 1], 1);
			} else {
				atomicSub(&b.free_spots[0], 1);
				atomicSub(&b.free_spots[1], 1);
			}
			b.to_play = to_play;
			b.moves++;
			// printf("Z %d\n", move);
			break;
		case -S:
		case -1:
		case 1:
		case S:
			if (b.stone[threadIdx.x] == to_play)
				group_merge[atomicAdd(&group_merge_n, 1)] = b.group[threadIdx.x];
			else if (b.stone[threadIdx.x] == S_NONE)
				atomicAdd(&b.libs[move], 1);
			break;
	}
	__syncthreads();

	/** Merge multiple groups if applicable */
	if (group_merge_n > 1) {
		if (threadIdx.x == move)
			b.group[threadIdx.x] = group_merge[0];
		else if (threadIdx.x == group_merge[0])
			b.libs[threadIdx.x] = 0;

		for (int i = 1; i < group_merge_n; i++)
			if (b.group[threadIdx.x] == group_merge[i])
				b.group[threadIdx.x] = group_merge[0];
		__syncthreads();

		/* Recalculate liberties */
		if (b.stone[threadIdx.x] == S_NONE
		    && (b.group[TU] == group_merge[0]
		        || b.group[TD] == group_merge[0]
		        || b.group[TL] == group_merge[0]
		        || b.group[TR] == group_merge[0]))
			atomicAdd(&b.libs[group_merge[0]], 1);

	/** otherwise just join the group and survey bonus libs for the group */
	} else if (group_merge_n == 1) {
		switch (move - (int)threadIdx.x) {
			case 0:
				b.group[threadIdx.x] = group_merge[0];
				break;
			case -S:
				if (b.stone[threadIdx.x] != S_NONE)
					break;
				if (b.group[TD] != group_merge[0]
				    && b.group[TL] != group_merge[0]
				    && b.group[TR] != group_merge[0])
					atomicAdd(&b.libs[group_merge[0]], 1);
				break;
			case -1:
				if (b.stone[threadIdx.x] != S_NONE)
					break;
				if (b.group[TU] != group_merge[0]
				    && b.group[TD] != group_merge[0]
				    && b.group[TR] != group_merge[0])
					atomicAdd(&b.libs[group_merge[0]], 1);
				break;
			case 1:
				if (b.stone[threadIdx.x] != S_NONE)
					break;
				if (b.group[TU] != group_merge[0]
				    && b.group[TD] != group_merge[0]
				    && b.group[TL] != group_merge[0])
					atomicAdd(&b.libs[group_merge[0]], 1);
				break;
			case S:
				if (b.stone[threadIdx.x] != S_NONE)
					break;
				if (b.group[TU] != group_merge[0]
				    && b.group[TL] != group_merge[0]
				    && b.group[TR] != group_merge[0])
					atomicAdd(&b.libs[group_merge[0]], 1);
				break;
		}

	/** or create a new group! */
	} else {
		if (threadIdx.x == move)
			b.group[threadIdx.x] = threadIdx.x;
	}

	__syncthreads();

	/* Both following capture tests propagate liberties themselves */

	/** Take out opponent's stones */
	if (b.group[threadIdx.x] > 0 && b.stone[threadIdx.x] == b.to_play) {
		capture_stones(&b);
	}
	__syncthreads();

	/** Take out our stones */
	if (b.group[threadIdx.x] > 0 && b.stone[threadIdx.x] != b.to_play) {
		capture_stones(&b);
	}
	__syncthreads();

	/** Propagate liberties */
	if (b.group[threadIdx.x] > 0) {
		b.libs[threadIdx.x] = b.libs[b.group[threadIdx.x]];
	}
	__syncthreads();

	/** Check if we are an eye */
	if (b.stone[threadIdx.x] == S_NONE) {
		survey_eye(b);
	}
	__syncthreads();

	/** Update probabilities */
	if (b.free_spots[b.to_play - 1] > 0) {
		calc_probability(b);
		__syncthreads();
	}

	if (!threadIdx.x)
		dprint_board(&b);
}

__device__ void
board2board(struct board *b1, struct board *b2)
{
	/* First thread loads global state */
	if (!threadIdx.x) {
		b2->random = b1->random;
		b2->free_spots[0] = b1->free_spots[0];
		b2->free_spots[1] = b1->free_spots[1];
		b2->to_play = b1->to_play;
		b2->moves = b1->moves;
		b2->komi = b1->komi;
	}
	/* Then each thread loads one element */
	b2->stone[threadIdx.x] = b1->stone[threadIdx.x];
	b2->group[threadIdx.x] = b1->group[threadIdx.x];
	b2->libs[threadIdx.x] = b1->libs[threadIdx.x];
	b2->p[threadIdx.x] = b1->p[threadIdx.x];
}

__device__ void
count_score(struct board &b, int &score)
{
	/* XXX: This is horribly ineffective. */
	__shared__ int black, white;
	if (!threadIdx.x) {
		black = white = 0;
	}
	__syncthreads();
	switch (b.stone[threadIdx.x]) {
		case S_BLACK: atomicAdd(&black, 1); break;
		case S_WHITE: atomicAdd(&white, 1); break;
		case S_NONE:
			if (b.stone[TL] == S_BLACK || b.stone[TR] == S_BLACK)
				atomicAdd(&black, 1);
			else
				atomicAdd(&white, 1);
			break;
	}
	__syncthreads();
	if (!threadIdx.x)
		score = b.komi + white - black;
}

__global__ void
player(struct board *gb, int *score)
{
	/** First, load board into shared memory */
	__shared__ struct board b;
	board2board(&gb[blockIdx.x], &b);

	__syncthreads();

	/** Play the game */

#if 0
	/* For device code debugging - run fixed number of iterations */
	for (int i = 0; i < 128; i++) {
#else
	while (b.moves < MAX_MOVES && b.free_spots[0] + b.free_spots[1] > 0) {
#endif
		if (b.free_spots[b.to_play - 1] > 0) {
			play_one_move(b);
		} else {
			/* pass and let the other player make a move */
			if (!threadIdx.x) {
				b.moves++;
				b.to_play = b.to_play == S_BLACK ? S_WHITE : S_BLACK;
			}
			__syncthreads();
			calc_probability(b);
			__syncthreads();
		}
	}

	/** Count score */
	count_score(b, score[blockIdx.x]);

	/** Send board back */
	board2board(&b, &gb[blockIdx.x]);
}


void
print_board(struct board *bp)
{
	for (int i = 0; i < S; i++) {
		int j;
		for (j = 0; j < S; j++) {
			int st = bp->stone[i * S + j];
			printf("%c ", st == S_EDGE ? '#' : st == S_WHITE ? 'O' : st == S_BLACK ? 'X' : '.');
		}
		printf(" ");
		for (j = 0; j < S; j++)
			printf("%03d ", bp->group[i * S + j]);
		printf(" ");
		for (j = 0; j < S; j++)
			printf("%02d ", bp->libs[i * S + j]);
#if 0
		printf(" ");
		for (j = 0; j < S; j++)
			printf("%1.02f ", bp->p[i * S + j]);
#endif
		printf("\n");
	}
	printf("random %d moves %d free_spots %d,%d to_play %d komi %d\n", bp->random, bp->moves, bp->free_spots[0], bp->free_spots[1], bp->to_play, bp->komi);
}

clock_t start_time;
void timestats(void) {
	struct tms t;
	clock_t now = times(&t);
	int u = sysconf(_SC_CLK_TCK);
	printf("TIMES: user %fs, system %fs, total %fs\n",
			(float)t.tms_utime / u,
			(float)t.tms_stime / u,
			(float)(now - start_time) / u);
}

int
main(int argc, char *argv[])
{
	if (argc < 3) {
		fprintf(stderr, "Usage: %s RANDSEED PLAYOUTS PLAYOUTSPERJOB\n", argv[0]);
		exit(EXIT_FAILURE);
	}
	b.random = atoi(argv[1]);
	int iters = atoi(argv[2]), ppj = atoi(argv[3]);
	b.free_spots[0] = b.free_spots[1] = (S - 2) * (S - 2);
	b.moves = b.komi = 0;
	b.to_play = S_BLACK;
	for (int i = 0; i < S2; i++) {
		b.stone[i] = (i % S == 0 || i % S == S-1 || i / S == 0 || i / S == S-1) ? S_EDGE : S_NONE;
		b.group[i] = 0;
		b.libs[i] = 0;
		if (b.stone[i] == S_NONE)
			b.p[i] = 1.f / b.free_spots[b.to_play - 1];
		else
			b.p[i] = 0;
	}
	//print_board(&b);

	start_time = times(NULL);

	int score[ppj];
	struct board *gb; int *gscore;
	hipMalloc((void**) &gb, sizeof(*gb) * ppj);
	hipMalloc((void**) &gscore, sizeof(*gscore) * ppj);

	int black = 0, white = 0;

	for (int i = 0; i < iters; i += ppj) {
		//printf("Copying boards to GPU...\n");
		for (int j = 0; j < ppj; j++) {
			hipMemcpy(&gb[j], &b, sizeof(b), hipMemcpyHostToDevice);
			b.random++;
		}

		int blocks = ppj;
		int threads = S2;
		//timestats();
		//printf("Crunching...\n");
		player <<< blocks, threads >>> (gb, gscore);
		//timestats();

		//printf("Copying score back...\n");
		hipMemcpy(&score, gscore, sizeof(*gscore) * ppj, hipMemcpyDeviceToHost);
		for (int j = 0; j < ppj; j++)
			if (score[j] > 0)
				white++;
			else if (score[j] < 0)
				black++;
#if 0
		for (int j = 0; j < ppj; j++)
			printf("%d ", score[j]);
		printf("\n");
#endif
#if 0
		struct board b0;
		hipMemcpy(&b0, &gb[0], sizeof(b0), hipMemcpyDeviceToHost);
		print_board(&b0);
#endif
		//timestats();
	}

	hipFree(gb);
	hipFree(gscore);
	hipDeviceReset();
	timestats();
	printf("Win stats: %.4f%% for black (%d games)\n", (float)black/(black+white), black);
	return EXIT_SUCCESS;
}
